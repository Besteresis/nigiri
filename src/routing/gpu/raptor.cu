#include "hip/hip_runtime.h"
#include "nigiri/routing/gpu/raptor.h"

#include <cstdio>
#include <iostream>

#include "cuda/std/array"
#include "cuda/std/span"

#include "hip/hip_cooperative_groups.h"

#include "thrust/device_vector.h"
#include "thrust/fill.h"
#include "thrust/host_vector.h"

#include "nigiri/routing/gpu/device_timetable.cuh"
#include "nigiri/routing/gpu/raptor_impl.cuh"
#include "nigiri/routing/gpu/types.cuh"

namespace cg = cooperative_groups;

namespace nigiri::routing::gpu {

#define CUDA_CHECK(code)                                       \
  utl_verify((code) == hipSuccess, "CUDA error: {} at {}:{}", \
             hipGetErrorString(code), __FILE__, __LINE__);

struct gpu_timetable::impl {
  using t = timetable;

  explicit impl(timetable const& tt)
      : n_locations_{tt.n_locations()},
        n_routes_{tt.n_routes()},
        transfer_time_{to_device(tt.locations_.transfer_time_)},
        footpaths_out_{tt.locations_.footpaths_out_[0]},
        footpaths_in_{tt.locations_.footpaths_in_[0]},
        route_stop_times_{to_device(tt.route_stop_times_)},
        route_stop_time_ranges_{to_device(tt.route_stop_time_ranges_)},
        route_transport_ranges_{to_device(tt.route_transport_ranges_)},
        route_clasz_{to_device(tt.route_clasz_)},
        route_bikes_allowed_{to_device(tt.route_bikes_allowed_)},
        route_bikes_allowed_per_section_{tt.route_bikes_allowed_per_section_},
        route_location_seq_{tt.route_location_seq_},
        location_routes_{tt.location_routes_},
        transport_traffic_days_{to_device(tt.transport_traffic_days_)},
        bitfields_{to_device(tt.bitfields_)},
        internal_interval_days_{tt.internal_interval_days()} {}

  device_timetable to_device_timetable() const {
    return {.n_locations_ = n_locations_,
            .n_routes_ = n_routes_,
            .transfer_time_ = transfer_time_,
            .footpaths_out_ = to_view(footpaths_out_),
            .footpaths_in_ = to_view(footpaths_in_),
            .route_stop_times_ = to_view(route_stop_times_),
            .route_stop_time_ranges_ = to_view(route_stop_time_ranges_),
            .route_transport_ranges_ = to_view(route_transport_ranges_),
            .route_clasz_ = to_view(route_clasz_),
            .route_bikes_allowed_ = {to_view(route_bikes_allowed_)},
            .route_bikes_allowed_per_section_ =
                to_view(route_bikes_allowed_per_section_),
            .route_location_seq_ = to_view(route_location_seq_),
            .location_routes_ = to_view(location_routes_),
            .transport_traffic_days_ = to_view(transport_traffic_days_),
            .bitfields_ = to_view(bitfields_),
            .internal_interval_days_ = internal_interval_days_};
  }

  std::uint32_t n_locations_;
  std::uint32_t n_routes_;

  thrust::device_vector<u8_minutes> transfer_time_;
  device_vecvec<decltype(t{}.locations_.footpaths_out_[0])> footpaths_out_;
  device_vecvec<decltype(t{}.locations_.footpaths_in_[0])> footpaths_in_;

  thrust::device_vector<delta> route_stop_times_;
  thrust::device_vector<interval<std::uint32_t>> route_stop_time_ranges_;
  thrust::device_vector<interval<transport_idx_t>> route_transport_ranges_;
  thrust::device_vector<clasz> route_clasz_;
  thrust::device_vector<std::uint64_t> route_bikes_allowed_;
  device_vecvec<decltype(t{}.route_bikes_allowed_per_section_)>
      route_bikes_allowed_per_section_;

  device_vecvec<decltype(t{}.route_location_seq_)> route_location_seq_;
  device_vecvec<decltype(t{}.location_routes_)> location_routes_;

  thrust::device_vector<bitfield_idx_t> transport_traffic_days_;
  thrust::device_vector<bitfield> bitfields_;

  interval<date::sys_days> internal_interval_days_;
};

gpu_timetable::gpu_timetable(timetable const& tt)
    : impl_{std::make_unique<impl>(tt)} {}

gpu_timetable::~gpu_timetable() = default;

struct gpu_raptor_state::impl {
  explicit impl(gpu_timetable const& gtt)
      : tt_{gtt.impl_->to_device_timetable()} {}

  void resize(unsigned n_locations,
              unsigned n_routes,
              unsigned n_rt_transports,
              std::array<nigiri::bitvec, kMaxVias> const& is_via,
              std::vector<via_stop> const& via_stops,
              nigiri::bitvec const& is_dest,
              std::vector<std::uint16_t> const& dist_to_dest,
              std::vector<std::uint16_t> const& lb) {
    is_intermodal_dest_ = !dist_to_dest.empty();
    n_locations_ = host_state_.n_locations_ = n_locations;

    time_at_dest_.resize(kMaxTransfers + 1);

    tmp_storage_.resize(n_locations * (kMaxVias + 1));
    best_storage_.resize(n_locations * (kMaxVias + 1));
    round_times_storage_.resize(n_locations * (kMaxVias + 1) *
                                (kMaxTransfers + 1));
    host_round_times_.resize(round_times_storage_.size());
    station_mark_.resize(n_locations);
    prev_station_mark_.resize(n_locations);
    route_mark_.resize(n_routes);
    rt_transport_mark_.resize(n_rt_transports);
    end_reachable_.resize(n_locations);

    for (auto i = 0U; i != is_via.size(); ++i) {
      is_via_[i].resize(is_via[i].blocks_.size());
      thrust::copy(thrust::cuda::par_nosync, begin(is_via[i].blocks_),
                   end(is_via[i].blocks_), begin(is_via_[i]));
    }

    via_stops_.resize(via_stops.size());
    thrust::copy(thrust::cuda::par_nosync, begin(via_stops), end(via_stops),
                 begin(via_stops_));

    is_dest_.resize(is_dest.blocks_.size());
    utl::verify(
        hipSuccess ==
            hipMemcpy(thrust::raw_pointer_cast(is_dest_.data()),
                       is_dest.blocks_.data(),
                       is_dest.blocks_.size() *
                           sizeof(std::decay_t<decltype(is_dest)>::block_t),
                       hipMemcpyHostToDevice),
        "could not copy is_dest bitvector");

    dist_to_dest_.resize(dist_to_dest.size());
    thrust::copy(thrust::cuda::par_nosync, begin(dist_to_dest),
                 end(dist_to_dest), begin(dist_to_dest_));

    lb_.resize(lb.size());
    utl::verify(
        hipSuccess == hipMemcpy(thrust::raw_pointer_cast(lb_.data()),
                                  lb.data(), lb.size() * sizeof(std::uint16_t),
                                  hipMemcpyHostToDevice),
        "could not copy lb bitvector");
  }

  template <via_offset_t Vias>
  cuda::std::span<cuda::std::array<delta_t, Vias + 1>> get_tmp() {
    return {reinterpret_cast<cuda::std::array<delta_t, Vias + 1>*>(
                thrust::raw_pointer_cast(tmp_storage_.data())),
            n_locations_};
  }

  template <via_offset_t Vias>
  cuda::std::span<cuda::std::array<delta_t, Vias + 1>> get_best() {
    return {reinterpret_cast<cuda::std::array<delta_t, Vias + 1>*>(
                thrust::raw_pointer_cast(best_storage_.data())),
            n_locations_};
  }

  template <via_offset_t Vias>
  cuda::std::span<cuda::std::array<delta_t, Vias + 1> const> get_best() const {
    return {reinterpret_cast<cuda::std::array<delta_t, Vias + 1> const*>(
                thrust::raw_pointer_cast(best_storage_.data())),
            n_locations_};
  }

  template <via_offset_t Vias>
  device_flat_matrix_view<cuda::std::array<delta_t, Vias + 1>>
  get_round_times() {
    return {{reinterpret_cast<cuda::std::array<delta_t, Vias + 1>*>(
                 thrust::raw_pointer_cast(round_times_storage_.data())),
             n_locations_ * (kMaxTransfers + 1)},
            kMaxTransfers + 1U,
            n_locations_};
  }

  template <via_offset_t Vias>
  device_flat_matrix_view<cuda::std::array<delta_t, Vias + 1> const>
  get_round_times() const {
    return {{reinterpret_cast<cuda::std::array<delta_t, Vias + 1> const*>(
                 thrust::raw_pointer_cast(round_times_storage_.data())),
             n_locations_ * (kMaxTransfers + 1)},
            kMaxTransfers + 1U,
            n_locations_};
  }

  unsigned n_locations_;
  bool is_intermodal_dest_;
  thrust::device_vector<delta_t> time_at_dest_;
  thrust::device_vector<delta_t> tmp_storage_;
  thrust::device_vector<delta_t> best_storage_;
  thrust::device_vector<delta_t> round_times_storage_;
  thrust::device_vector<std::uint32_t> station_mark_;
  thrust::device_vector<std::uint32_t> prev_station_mark_;
  thrust::device_vector<std::uint32_t> route_mark_;
  thrust::device_vector<std::uint32_t> rt_transport_mark_;

  thrust::device_vector<std::uint32_t> end_reachable_;
  thrust::device_vector<std::uint64_t> is_dest_;
  std::array<thrust::device_vector<std::uint64_t>, kMaxVias> is_via_;
  thrust::device_vector<via_stop> via_stops_;
  thrust::device_vector<std::uint16_t> dist_to_dest_;
  thrust::device_vector<std::uint16_t> lb_;

  device_timetable tt_;

  thrust::host_vector<delta_t> host_round_times_;
  raptor_state host_state_;
};

gpu_raptor_state::gpu_raptor_state(gpu_timetable const& gtt)
    : impl_{std::make_unique<impl>(gtt)} {}

gpu_raptor_state::~gpu_raptor_state() = default;

template <direction SearchDir, bool Rt, via_offset_t Vias>
gpu_raptor<SearchDir, Rt, Vias>::gpu_raptor(
    timetable const& tt,
    rt_timetable const* rtt,
    gpu_raptor_state& state,
    nigiri::bitvec& is_dest,
    std::array<nigiri::bitvec, kMaxVias> const& is_via,
    std::vector<std::uint16_t> const& dist_to_dest,
    hash_map<location_idx_t, std::vector<td_offset>> const& td_dist_to_dest,
    std::vector<std::uint16_t> const& lb,
    std::vector<via_stop> const& via_stops,
    day_idx_t const base,
    clasz_mask_t const allowed_claszes,
    bool const require_bike_transport,
    bool const is_wheelchair,
    transfer_time_settings const& tts)
    : tt_{tt},
      rtt_{rtt},
      n_days_{tt_.internal_interval_days().size().count()},
      n_locations_{tt_.n_locations()},
      n_routes_{tt.n_routes()},
      n_rt_transports_{Rt ? rtt->n_rt_transports() : 0U},
      state_{state},
      is_dest_{is_dest},
      is_via_{is_via},
      dist_to_end_{dist_to_dest},
      td_dist_to_end_{td_dist_to_dest},
      lb_{lb},
      via_stops_{via_stops},
      base_{base},
      allowed_claszes_{allowed_claszes},
      require_bike_transport_{require_bike_transport},
      is_wheelchair_{is_wheelchair},
      transfer_time_settings_{tts} {
  state_.impl_->resize(tt.n_locations(), tt.n_routes(),
                       rtt ? rtt->n_rt_transports() : 0U, is_via, via_stops,
                       is_dest, dist_to_dest, lb);
  reset_arrivals();
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
__global__ void exec_raptor(unixtime_t const start_time,
                            std::uint8_t const max_transfers,
                            unixtime_t const worst_time_at_dest,
                            raptor_impl<SearchDir, Rt, Vias> r) {
  r.execute(start_time, max_transfers, worst_time_at_dest);
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
void gpu_raptor<SearchDir, Rt, Vias>::execute(
    unixtime_t const start_time,
    std::uint8_t const max_transfers,
    unixtime_t const worst_time_at_dest,
    profile_idx_t,
    pareto_set<journey>& results) {
  auto const starts =
      thrust::device_vector<std::pair<location_idx_t, unixtime_t>>{starts_};
  hipDeviceSynchronize();
  CUDA_CHECK(hipPeekAtLastError());

  auto& s = *state_.impl_;
  exec_raptor<SearchDir, Rt, Vias><<<1, 1>>>(
      start_time, max_transfers, worst_time_at_dest,
      raptor_impl<SearchDir, Rt, Vias>{
          .tt_ = s.tt_,
          .n_locations_ = s.tt_.n_locations_,
          .n_routes_ = s.tt_.n_routes_,
          .n_rt_transports_ = 0U,  // TODO
          .transfer_time_settings_ = transfer_time_settings_,
          .max_transfers_ = max_transfers,
          .allowed_claszes_ = allowed_claszes_,
          .require_bike_transport_ = require_bike_transport_,
          .base_ = base_,
          .worst_time_at_dest_ = worst_time_at_dest,
          .is_intermodal_dest_ = s.is_intermodal_dest_,
          .starts_ = to_view(starts),
          .is_via_ =
              [&]() {
                auto ret = cuda::std::array<device_bitvec<std::uint64_t const>,
                                            kMaxVias>{};
                for (auto i = 0U; i != kMaxVias; ++i) {
                  ret[i] = {.blocks_ = to_view(s.is_via_[i])};
                }
                return ret;
              }(),
          .via_stops_ = to_view(s.via_stops_),
          .is_dest_ = {to_view(s.is_dest_)},
          .end_reachable_ = {to_mutable_view(s.end_reachable_)},
          .dist_to_end_ = to_view(s.dist_to_dest_),
          .lb_ = to_view(s.lb_),
          .round_times_ = s.get_round_times<Vias>(),
          .best_ = s.get_best<Vias>(),
          .tmp_ = s.get_tmp<Vias>(),
          .time_at_dest_ = to_mutable_view(s.time_at_dest_),
          .station_mark_ = {to_mutable_view(s.station_mark_)},
          .prev_station_mark_ = {to_mutable_view(s.prev_station_mark_)},
          .route_mark_ = {to_mutable_view(s.route_mark_)}});

  hipDeviceSynchronize();
  CUDA_CHECK(hipPeekAtLastError());

  utl::verify(
      hipSuccess ==
          hipMemcpy(thrust::raw_pointer_cast(s.host_round_times_.data()),
                     thrust::raw_pointer_cast(s.round_times_storage_.data()),
                     s.round_times_storage_.size() * sizeof(delta_t),
                     hipMemcpyDeviceToHost),
      "could not copy is_dest bitvector");

  sync_round_times();

  auto const round_times = s.host_state_.get_round_times<Vias>();
  auto const end_k = std::min(max_transfers, kMaxTransfers) + 1U;
  is_dest_.for_each_set_bit([&](auto const i) {
    for (auto k = 1U; k != end_k; ++k) {
      auto const dest_time = round_times[k][i][Vias];
      if (dest_time != kInvalid) {
        auto const [optimal, it, dominated_by] = results.add(
            journey{.legs_ = {},
                    .start_time_ = start_time,
                    .dest_time_ = delta_to_unix(base(), dest_time),
                    .dest_ = location_idx_t{i},
                    .transfers_ = static_cast<std::uint8_t>(k - 1)});
      }
    }
  });
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
void gpu_raptor<SearchDir, Rt, Vias>::sync_round_times() {
  auto& s = *state_.impl_;
  s.host_state_.round_times_storage_.resize(s.host_round_times_.size());
  std::copy(begin(s.host_round_times_), end(s.host_round_times_),
            begin(s.host_state_.round_times_storage_));
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
void gpu_raptor<SearchDir, Rt, Vias>::reset_arrivals() {
  thrust::fill(thrust::cuda::par_nosync, begin(state_.impl_->time_at_dest_),
               end(state_.impl_->time_at_dest_), kInvalid);
  thrust::fill(thrust::cuda::par_nosync,
               begin(state_.impl_->round_times_storage_),
               end(state_.impl_->round_times_storage_), kInvalid);
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
void gpu_raptor<SearchDir, Rt, Vias>::next_start_time() {
  starts_.clear();
  thrust::fill(thrust::cuda::par_nosync, begin(state_.impl_->best_storage_),
               end(state_.impl_->best_storage_), kInvalid);
  thrust::fill(thrust::cuda::par_nosync, begin(state_.impl_->tmp_storage_),
               end(state_.impl_->tmp_storage_), kInvalid);
  thrust::fill(thrust::cuda::par_nosync,
               begin(state_.impl_->prev_station_mark_),
               end(state_.impl_->prev_station_mark_), 0U);
  thrust::fill(thrust::cuda::par_nosync, begin(state_.impl_->station_mark_),
               end(state_.impl_->station_mark_), 0U);
  thrust::fill(thrust::cuda::par_nosync, begin(state_.impl_->route_mark_),
               end(state_.impl_->route_mark_), 0U);
  if constexpr (Rt) {
    thrust::fill(thrust::cuda::par_nosync,
                 begin(state_.impl_->rt_transport_mark_),
                 end(state_.impl_->rt_transport_mark_), 0U);
  }
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
void gpu_raptor<SearchDir, Rt, Vias>::reconstruct(query const& q, journey& j) {
  reconstruct_journey<SearchDir>(tt_, rtt_, q, state_.impl_->host_state_, j,
                                 base(), base_);
}

template <direction SearchDir, bool Rt, via_offset_t Vias>
void gpu_raptor<SearchDir, Rt, Vias>::add_start(location_idx_t const l,
                                                unixtime_t const t) {
  starts_.emplace_back(l, t);
}

template class gpu_raptor<direction::kForward, true, 0U>;
template class gpu_raptor<direction::kForward, true, 1U>;
template class gpu_raptor<direction::kForward, true, 2U>;
template class gpu_raptor<direction::kForward, false, 0U>;
template class gpu_raptor<direction::kForward, false, 1U>;
template class gpu_raptor<direction::kForward, false, 2U>;
template class gpu_raptor<direction::kBackward, true, 0U>;
template class gpu_raptor<direction::kBackward, true, 1U>;
template class gpu_raptor<direction::kBackward, true, 2U>;
template class gpu_raptor<direction::kBackward, false, 0U>;
template class gpu_raptor<direction::kBackward, false, 1U>;
template class gpu_raptor<direction::kBackward, false, 2U>;

}  // namespace nigiri::routing::gpu